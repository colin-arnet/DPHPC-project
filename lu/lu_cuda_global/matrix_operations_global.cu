#include "hip/hip_runtime.h"
// reference:
// https://techbird.wordpress.com/2012/07/30/calling-cuda-program-from-cc-project/


// TRANSPOSE FROM: 
// TILE_SIZE = Warp size
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>


#include "matrix_util.h"
#include "matrix_operations_seq.h"

#define DOUBLE_PRECISION
#ifdef DOUBLE_PRECISION
using Scalar = double;
#else
using Scalar = float;
#endif


__global__ void add( Scalar *a, Scalar *b, Scalar *c) {
    int tid = blockIdx.x;	//Handle the data at the index
	c[tid] = a[tid] + b[tid];
}


__global__ void scale(Scalar *a, int size, int index){
 	int i;
	int start=(index*size+index);
	int end=(index*size+size);
	
	for(i=start+1;i<end;i++){
		a[i]=(a[i]/a[start]);
	}

}

__global__ void reduce(Scalar *a, int size, int index){
	int i;
       // int tid=threadIdx.x;
	int tid=blockIdx.x;
	int start= ((index+tid+1)*size+index);
	int end= ((index+tid+1)*size+size);
    for(i=start+1;i<end;i++){
        // a[i]=a[i]-(a[start]*a[(index*size)+i]);
	    a[i]=a[i]-(a[start]*a[(index*size)+(index+(i-start))]);
    }

}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void global_lu(int N, Scalar* a){

    Scalar *dev_a;
    if ( hipSuccess != hipMalloc ( (void**)&dev_a, N*N* sizeof (Scalar) )) {
        printf( "Error allocating memory on device!\n" );
    }
    gpuErrchk(hipMemcpy( dev_a, a, N*N*sizeof(Scalar), hipMemcpyHostToDevice));//copy array to device memory

    int i;
    for(i=0;i<N;i++){
        scale<<<1,1>>>(dev_a, N, i);
        reduce<<<(N-i-1),1>>>(dev_a, N, i);
    }
    hipMemcpy( a, dev_a, N*N*sizeof(Scalar),hipMemcpyDeviceToHost );//copy array back to host
}
