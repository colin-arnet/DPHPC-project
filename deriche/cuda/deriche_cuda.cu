#include "hip/hip_runtime.h"
/**
 * This version is stamped on May 10, 2016
 *
 * Contact:
 *   Louis-Noel Pouchet <pouchet.ohio-state.edu>
 *   Tomofumi Yuki <tomofumi.yuki.fr>
 *
 * Web address: http://polybench.sourceforge.net
 */
/* deriche.c: this file is part of PolyBench/C */


#ifdef DOUBLE_PRECISION
typedef double DATA_TYPE;
#else
typedef float DATA_TYPE;
#endif

#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <stdio.h>

/* Include polybench common header. */
#include "polybench.h"

/* Include benchmark-specific header. */
#include "deriche.h"

#define BLOCK_SIZE 16


__global__ void gpu_deriche_left_right(int w,int h, DATA_TYPE alpha, DATA_TYPE *imgIn, DATA_TYPE* imgOut, DATA_TYPE *y1, DATA_TYPE *y2)
{ 
    int j;
    
    int blockId= blockIdx.y * gridDim.x + blockIdx.x;
    int row = blockId * blockDim.x + threadIdx.x;


    DATA_TYPE xm1, ym1, ym2;
    DATA_TYPE xp1, xp2;
    DATA_TYPE yp1, yp2;

    DATA_TYPE k;
    DATA_TYPE a1, a2, a3, a4; //, a5, a6, a7, a8;
    DATA_TYPE b1, b2, c1;

    k = (SCALAR_VAL(1.0)-EXP_FUN(-alpha))*(SCALAR_VAL(1.0)-EXP_FUN(-alpha))/(SCALAR_VAL(1.0)+SCALAR_VAL(2.0)*alpha*EXP_FUN(-alpha)-EXP_FUN(SCALAR_VAL(2.0)*alpha));
    a1 = k;
    a2 = k*EXP_FUN(-alpha)*(alpha-SCALAR_VAL(1.0));
    a3 = k*EXP_FUN(-alpha)*(alpha+SCALAR_VAL(1.0));
    a4 = -k*EXP_FUN(SCALAR_VAL(-2.0)*alpha);
    b1 =  POW_FUN(SCALAR_VAL(2.0),-alpha);
    b2 = -EXP_FUN(SCALAR_VAL(-2.0)*alpha);
    c1 = 1;
    
    if(row < h) 
    {
        ym1 = SCALAR_VAL(0.0);
        ym2 = SCALAR_VAL(0.0);
        xm1 = SCALAR_VAL(0.0);
        for (j=0; j<w; j++) {
            y1[row*w + j] = a1*imgIn[row*w +j] + a2*xm1 + b1*ym1 + b2*ym2;
            xm1 = imgIn[row*w +j];
            ym2 = ym1;
            ym1 = y1[row*w +j];
        }

        yp1 = SCALAR_VAL(0.0);
        yp2 = SCALAR_VAL(0.0);
        xp1 = SCALAR_VAL(0.0);
        xp2 = SCALAR_VAL(0.0);
        for (j=w-1; j>=0; j--) {
            y2[row*w +j] = a3*xp1 + a4*xp2 + b1*yp1 + b2*yp2;
            xp2 = xp1;
            xp1 = imgIn[row*w +j];
            yp2 = yp1;
            yp1 = y2[row*w +j];
            imgOut[row*w +j] = c1 * (y1[row*w +j] + y2[row*w +j]);
        }
    
    }


} 

__global__ void gpu_deriche_up_down(int w,int h, DATA_TYPE alpha, DATA_TYPE *imgIn, DATA_TYPE* imgOut, DATA_TYPE *y1, DATA_TYPE *y2)
{ 
    int i;
    
    int blockId= blockIdx.y * gridDim.x + blockIdx.x;
    int col = blockId * blockDim.x + threadIdx.x;
    
    DATA_TYPE  tm1, ym1, ym2;
    DATA_TYPE tp1, tp2;
    DATA_TYPE yp1, yp2;

    DATA_TYPE k;
    DATA_TYPE  a5, a6, a7, a8;
    DATA_TYPE b1, b2, c2;

    k = (SCALAR_VAL(1.0)-EXP_FUN(-alpha))*(SCALAR_VAL(1.0)-EXP_FUN(-alpha))/(SCALAR_VAL(1.0)+SCALAR_VAL(2.0)*alpha*EXP_FUN(-alpha)-EXP_FUN(SCALAR_VAL(2.0)*alpha));
    a5 = k;
    a6 = k*EXP_FUN(-alpha)*(alpha-SCALAR_VAL(1.0));
    a7 = k*EXP_FUN(-alpha)*(alpha+SCALAR_VAL(1.0));
    a8 = -k*EXP_FUN(SCALAR_VAL(-2.0)*alpha);
    b1 =  POW_FUN(SCALAR_VAL(2.0),-alpha);
    b2 = -EXP_FUN(SCALAR_VAL(-2.0)*alpha);
    c2 = 1;
    
    if(col < w) 
    {
        tm1 = SCALAR_VAL(0.0);
        ym1 = SCALAR_VAL(0.0);
        ym2 = SCALAR_VAL(0.0);
        for (i=0; i<h; i++) {
            y1[i*w + col] = a5*imgOut[i*w + col] + a6*tm1 + b1*ym1 + b2*ym2;
            tm1 = imgOut[i*w + col];
            ym2 = ym1;
            ym1 = y1 [i*w + col];
        }


        tp1 = SCALAR_VAL(0.0);
        tp2 = SCALAR_VAL(0.0);
        yp1 = SCALAR_VAL(0.0);
        yp2 = SCALAR_VAL(0.0);
        for (i=h; i>=0; i--) {
            y2[i*w + col] = a7*tp1 + a8*tp2 + b1*yp1 + b2*yp2;
            tp2 = tp1;
            tp1 = imgOut[i*w + col];
            yp2 = yp1;
            yp1 = y2[i*w + col];
        }

        for(int i = 0; i < h; i++) {
            imgOut[i*w + col] = c2*(y1[i*w + col] + y2[i*w + col]);
        }
    }

} 

void wrapper_deriche (DATA_TYPE* imgIn_host, DATA_TYPE* imgOut_host, int w, int h, DATA_TYPE alpha) {

    // Device pointers
    DATA_TYPE *imgIn_device, *imgOut_device, *y1_device, *y2_device;

    hipError_t error;
    error = hipMalloc((void **) &imgIn_device, sizeof(DATA_TYPE)*w*h);
    if (error != hipSuccess) { 
        printf ("Memory allocation error imgIn, code: %d \n", error);
        hipFree(imgIn_device);
        return;
    }

    error = hipMalloc((void **) &imgOut_device, sizeof(DATA_TYPE)*w*h);
    if (error != hipSuccess) { 
        printf ("Memory allocation error imgOut, code: %d \n", error);
        hipFree(imgIn_device);
        hipFree(imgOut_device);
        return;
    }

    error = hipMalloc((void **) &y1_device, sizeof(DATA_TYPE)*w*h);
    if (error != hipSuccess) { 
        printf ("Memory allocation error y1_device, code: %d \n", error);
        hipFree(imgIn_device);
        hipFree(imgOut_device);
        hipFree(y1_device);
        return;
    }

    error = hipMalloc((void **) &y2_device, sizeof(DATA_TYPE)*w*h);
    if (error != hipSuccess) { 
        printf ("Memory allocation error y2_device, code: %d \n", error);
        hipFree(imgIn_device);
        hipFree(imgOut_device);
        hipFree(y1_device);
        hipFree(y2_device);
        return;
    }

    //rintf("Memory allocation succeedd!");
    hipMemcpy(imgIn_device, imgIn_host, sizeof(DATA_TYPE)*h*w, hipMemcpyHostToDevice);

    int num_threads = 1024;
    int num_blocks = (h + BLOCK_SIZE - 1) / BLOCK_SIZE;
    gpu_deriche_left_right<<<num_blocks, num_threads>>> (w, h, alpha, imgIn_device, imgOut_device, y1_device, y2_device);

    num_blocks = (w + BLOCK_SIZE - 1) / BLOCK_SIZE;
    gpu_deriche_up_down<<<num_blocks, num_threads>>> (w, h, alpha, imgOut_device, imgOut_device, y1_device, y2_device);

    hipMemcpy(imgOut_host, imgOut_device, sizeof(DATA_TYPE)*h*w, hipMemcpyDeviceToHost);

    hipFree(imgIn_device);
    hipFree(imgOut_device);
    hipFree(y1_device);
    hipFree(y2_device);

}

