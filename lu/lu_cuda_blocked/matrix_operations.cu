#include "hip/hip_runtime.h"
// reference:
// https://techbird.wordpress.com/2012/07/30/calling-cuda-program-from-cc-project/


// TRANSPOSE FROM: 
// TILE_SIZE = Warp size
#include <stdio.h>
#include "matrix_util.h"
#include "matrix_operations_seq.h"

#define BLOCK_DIM 32

#define DOUBLE_PRECISION
#ifdef DOUBLE_PRECISION
using Scalar = double;
#else
using Scalar = float;
#endif

// CUDA kernels
__global__ void transpose_kernel(const Scalar* A, Scalar* A_trans, const int height, const int width){
    // shared block
	__shared__ Scalar block[BLOCK_DIM][BLOCK_DIM];

    // compute index in original matrix
    int x = blockIdx.x * BLOCK_DIM + threadIdx.x;
    int y = blockIdx.y * BLOCK_DIM + threadIdx.y;
    // put value into shared memory
    if(x < width && y < height){
        int index = y * width + x;
        block[threadIdx.y][threadIdx.x] = A[index];
    }
    // synchronize
    __syncthreads();
    // compute transposed index
    x = blockIdx.y * BLOCK_DIM + threadIdx.x;
    y = blockIdx.x * BLOCK_DIM + threadIdx.y;
    // put value from block to transposed matrix
    if(x < height && y < width){
        int index = y * height + x;
        A_trans[index] = block[threadIdx.x][threadIdx.y];
    }
}

__global__ void mat_mult_minus_kernel(const Scalar* A, const Scalar* B, Scalar* C, const int n, const int m, const int p)
{
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int myRow = threadIdx.y;
    int myCol = threadIdx.x;
    __shared__ Scalar As[BLOCK_DIM][BLOCK_DIM];
    __shared__ Scalar Bs[BLOCK_DIM][BLOCK_DIM];
    Scalar sum = C[row * p + col];
    for(int k = 0; k < (m / BLOCK_DIM) + 1; k++){
    	if(row < n && (k * BLOCK_DIM + myCol) < m){
	       As[myRow][myCol] = A[row * m + k * BLOCK_DIM + myCol];
	    } else {
	       As[myRow][myCol] = 0.0;
	    }
	    if((k * BLOCK_DIM + myRow) < m && col < p){
	       Bs[myRow][myCol] = B[(k * BLOCK_DIM + myRow) * p + col];
	    } else {
	       Bs[myRow][myCol] = 0.0;
	    }
	    __syncthreads();
	    for(int i = 0; i < BLOCK_DIM; i++){
	        sum -= As[myRow][i] * Bs[i][myCol];
	    }
	    __syncthreads();
    }
    if(row < n && col < p){
    	C[row * p + col] = sum;
    }
}

__global__ void trsm_kernel(const int n, const int m, const Scalar* L, Scalar* A){
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ Scalar block[];
    if(col < m){
    	for(int i = 0; i < n; i++){
		    block[i * BLOCK_DIM + threadIdx.x] = A[i * m + col];
        }
        __syncthreads();
        for(int i = 0; i < n; i++){
            Scalar value = 0.0;
            for(int j = 0; j < i; j++){
                value += L[i * n + j] * block[j * BLOCK_DIM + threadIdx.x];
            }
            block[i * BLOCK_DIM + threadIdx.x] -= value;
            A[i * m + col] -= value;
            __syncthreads();
        }
    }
}


__global__ void trans_trsm_kernel(const int n, const int m, const Scalar* L, Scalar* A){
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	extern __shared__ Scalar block[];
	if(row < n){
	    for(int i = 0; i < m; i++){
	        block[threadIdx.x * m + i] = A[row * m + i]; 
	    }
	    __syncthreads();
	    for(int i = 0; i < m; i++){
	       	Scalar value = 0.0;
		    for(int j = 0; j < i; j++){
			    value += L[j * m + i] * block[threadIdx.x * m + j];
		    }
            block[threadIdx.x * m + i] = (block[threadIdx.x * m + i] - value) / L[i * m + i];
		    A[row * m + i] = (block[threadIdx.x * m + i] - value) / L[i * m + i];
            __syncthreads();
	    }
	}
}




// CUDA wrappers
void wrapper_transpose(int n, int m, const Scalar* A, Scalar* A_trans)
{
    dim3 blocks((n / BLOCK_DIM) + 1, (m / BLOCK_DIM) + 1);
    dim3 threads(BLOCK_DIM, BLOCK_DIM);
    transpose_kernel<<<blocks, threads>>>(A, A_trans, n, m);

}

void wrapper_mat_mult_minus(int n, int m, int p, const Scalar* A, const Scalar* B, Scalar* result) 
{   
    dim3 blocks((n / BLOCK_DIM) + 1, (p / BLOCK_DIM) + 1);
    dim3 threads(BLOCK_DIM, BLOCK_DIM);
    printf("mat mult kernel blocks: %d; threads: %d\n", blocks.x * blocks.y, threads.x * threads.y);
    mat_mult_minus_kernel<<<blocks, threads>>>(A, B, result, n, m, p);
}


void wrapper_trsm(int n, int m, const Scalar* L, Scalar* A)
{
    // TODO: Call TRSM kernel
    // for each row
    dim3 blocks((m / BLOCK_DIM) + 1);
    dim3 threads(BLOCK_DIM);
    printf("trsm kernel blocks: %d; threads: %d\n", blocks.x, threads.x);
    trsm_kernel<<<blocks, threads, threads.x * n * sizeof(Scalar)>>>(n, m, L, A);
}   

void wrapper_trans_trsm(int n, int m, const Scalar* L, Scalar* A)
{
    // TODO: Call trans TRSM kernel
    // for each row
    dim3 blocks((n / BLOCK_DIM) + 1);
    dim3 threads(BLOCK_DIM);
    printf("trans trsm kernel blocks: %d; threads: %d\n", blocks.x, threads.x);
    trans_trsm_kernel<<<blocks, threads, threads.x * m * sizeof(Scalar)>>>(n, m, L, A);
}

void wrapper_lu(int n, Scalar* A, int bs){
    int already_done = 0;

    
    Scalar* A00 = (Scalar*) malloc(bs * sizeof(Scalar));
    Scalar* A01 = (Scalar*) malloc(bs * sizeof(Scalar));
    Scalar* A10 = (Scalar*) malloc(bs * sizeof(Scalar));
    Scalar* A11 = (Scalar*) malloc(bs * sizeof(Scalar));
    while (already_done < n) {
        if (already_done + bs >= n) {
            bs = n - already_done;
            if (bs == 1) {
                break;
            }
            Scalar* block = (Scalar*) malloc(bs * bs * sizeof(Scalar));
            extract_submatrix(already_done, already_done, bs, bs, n, A, block);
            lu_simple(bs, block);
            insert_submatrix(already_done, already_done, bs, bs, n, A, block);
	        free(block);
            break;
        }
        int A_n = n - already_done;

        A00 = (Scalar*) realloc(A00, bs * bs * sizeof(Scalar));
        A01 = (Scalar*) realloc(A01, bs * (A_n - bs) * sizeof(Scalar));
        A10 = (Scalar*) realloc(A10, (A_n - bs) * bs * sizeof(Scalar));
        A11 = (Scalar*) realloc(A11, (A_n - bs) * (A_n - bs) * sizeof(Scalar));

	    extract_submatrix(already_done, already_done, bs, bs, n, A, A00);
	    extract_submatrix(already_done, already_done + bs, bs, A_n - bs, n, A, A01);
        extract_submatrix(already_done + bs, already_done, A_n - bs, bs, n, A, A10);
        extract_submatrix(already_done + bs, already_done + bs, A_n - bs, A_n - bs, n, A, A11);
        
        lu_simple(bs, A00);

        Scalar* A00_d;
        hipMalloc((void**)&A00_d, bs * bs * sizeof(Scalar));
        Scalar* A01_d;
        hipMalloc((void**)&A01_d, bs * (A_n - bs) * sizeof(Scalar));
        Scalar* A10_d;
        hipMalloc((void**)&A10_d, (A_n - bs) * bs * sizeof(Scalar));
        Scalar* A11_d;
        hipMalloc((void**)&A11_d, (A_n - bs) * (A_n - bs) * sizeof(Scalar));


        hipMemcpy(A00_d, A00, bs * bs * sizeof(Scalar), hipMemcpyHostToDevice);
        hipMemcpy(A10_d, A10, bs * (A_n - bs) * sizeof(Scalar), hipMemcpyHostToDevice);
        hipMemcpy(A01_d, A01, bs * (A_n - bs) * sizeof(Scalar), hipMemcpyHostToDevice);
        hipMemcpy(A11_d, A11, (A_n - bs) * (A_n - bs) * sizeof(Scalar), hipMemcpyHostToDevice);
        
        wrapper_trsm(bs, A_n - bs, A00_d, A01_d);
	    wrapper_trans_trsm(A_n - bs, bs, A00_d, A10_d);
        wrapper_mat_mult_minus((A_n - bs), bs, (A_n - bs), A10_d, A01_d, A11_d);
	
       	hipMemcpy(A01, A01_d, bs * (A_n - bs) * sizeof(Scalar), hipMemcpyDeviceToHost);
        hipMemcpy(A10, A10_d, bs * (A_n - bs) * sizeof(Scalar), hipMemcpyDeviceToHost);
        hipMemcpy(A11, A11_d, (A_n - bs) * (A_n - bs) * sizeof(Scalar), hipMemcpyDeviceToHost);
	
        insert_submatrix(already_done, already_done, bs, bs, n, A, A00);
        insert_submatrix(already_done, already_done + bs, bs, A_n - bs, n, A, A01);
        insert_submatrix(already_done + bs, already_done, A_n - bs, bs, n, A, A10);
        insert_submatrix(already_done + bs, already_done + bs, A_n - bs, A_n - bs, n, A, A11);
	
	    hipFree(A00_d);
	    hipFree(A01_d);
	    hipFree(A10_d);
	    hipFree(A11_d);
	
        already_done += bs;
    }
    free(A00);
    free(A01);
    free(A10);
    free(A11);
}
