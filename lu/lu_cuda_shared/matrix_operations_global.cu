#include "hip/hip_runtime.h"
// reference:
// https://techbird.wordpress.com/2012/07/30/calling-cuda-program-from-cc-project/


// TRANSPOSE FROM: 
// TILE_SIZE = Warp size
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>


#include "matrix_util.h"
#include "matrix_operations_seq.h"

#define DOUBLE_PRECISION
#ifdef DOUBLE_PRECISION
using Scalar = double;
#else
using Scalar = float;
#endif


__global__ void add(Scalar *a, Scalar *b, Scalar *c) {
	int tid = blockIdx.x;	//Handle the data at the index
	c[tid] = a[tid] + b[tid];
}


__global__ void scale(Scalar *a, int size, int index){
 	int i;
	int start=(index*size+index);
	int end=(index*size+size);
	
	for(i=start+1;i<end;i++){
		a[i]=(a[i]/a[start]);
	}

}

__global__ void reduce(Scalar *a, int size, int index, int b_size){
	extern __shared__ float pivot[];
	int i;

	int tid=threadIdx.x;
	int bid=blockIdx.x;
	int block_size=b_size;

	int pivot_start=(index*size+index);
	int pivot_end=(index*size+size);

	int start;
	int end;
	int pivot_row;
	int my_row;

	if(tid==0){
		for(i=index;i<size;i++) pivot[i]=a[(index*size)+i];
	}

	__syncthreads();

	pivot_row=(index*size);
	my_row=(((block_size*bid) + tid)*size);
	start=my_row+index;
	end=my_row+size;

	if(my_row >pivot_row){
        for(i=start+1;i<end;i++){
            // a[i]=a[i]-(a[start]*a[(index*size)+i]);
			// a[i]=a[i]-(a[start]*a[(index*size)+(index+(i-start))]);
			a[i]=a[i]-(a[start]*pivot[(i-my_row)]);
        }
    }
}


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void global_lu(int N, Scalar* a){
	int blocks;

    Scalar *dev_a;
    if ( hipSuccess != hipMalloc ( (void**)&dev_a, N*N* sizeof (Scalar) )) {
        printf( "Error allocating memory on device!\n" );
    }
    gpuErrchk(hipMemcpy( dev_a, a, N*N*sizeof(Scalar), hipMemcpyHostToDevice));//copy array to device memory
    printf("%p \n", dev_a);

    int i;
	for(i=0;i<N;i++){
        scale<<<1,1>>>(dev_a,N,i);
        // blocks= ((N-i-1)/512)+1;
        blocks=((N/512));
        //	printf("Number of blocks rxd : %d \n",blocks);
        reduce<<<blocks,512,N*sizeof(float)>>>(dev_a,N,i,512);
      }
    
    hipMemcpy( a, dev_a, N*N*sizeof(Scalar),hipMemcpyDeviceToHost );//copy array back to host
}
